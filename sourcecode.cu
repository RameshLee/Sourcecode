#include "hip/hip_runtime.h"
#define Original_isGPU 1//1-Gpu, 0-cpu
#define Original_isFull 1 //1-full, 0-red

#define Original_isRev 0  //if(!full): then, 1-reverse(Dropoff first, then pickup),0-NotReverse.
#define SecondInsertionInCPU 1 //1-Yes, 0-No

// Run out of memory is possible due to two reasons: Expectation & Edge[2*n][2*n] array. Interactive gpu.
// NOTE: change TW in generate_samples() for each problem.

#define TotalRequests 120
#define TotalVehicles 11

#define Start_Window 200//330
#define End_Window 512//590

#define Str1 "./problems/general/pr05.txt"//a5-50.txt"
#define DeclaredProblem Str1
#define Defined_MaxTemperature 20

#define ExpectedScenarios 20
#define DegreeofDynamism 0.5 //(0,1)

#define Expectation 8500

#define TotalThreads 32 //max:1024
#define Expected_Blocks ((Expectation+T-1)/T)

#define Relocation_Strategy 0

#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <ctime>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <iterator>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "cuda_occupancy.h"
#include ""
#include "hip/hip_fp16.h"
#include <hip/hip_runtime.h>
#include <memory>
#include <random>
#define min(a,b) ((a) < (b) ? (a) : (b))
#define max(a,b) ((a) > (b) ? (a) : (b))
//#define RandomNumber(Min,Max) ((float(rand()) / float(RAND_MAX)) * (Max - Min)) + Min;
#define RandomNumber(min,max) rand() % (max - min + 1) + min;
#define RandomPick(b,a) ((b - a) * ((float)rand() / RAND_MAX)) + a //b=MAX, a=MIN // returns in-between numbers as well

//#include <hip/hip_cooperative_groups.h>
//using namespace cooperative_groups;

using namespace std;

#define n TotalRequests
#define m TotalVehicles
#define Var	2*(TotalRequests+TotalVehicles)
#define TotalNodes 2*n

//Class generation and Kernel call that maps __device__ function inside class to be called for every element of an array (threads)
//Initial commit
#define CHECK(r) {_check((r), __LINE__);}
#define Blocks 1
#define Threads 1
#define ExpectedPath 2*TotalRequests

#define w1 1//8
#define w2 0//3
#define w3 0//1
#define w4 0//1
#define w5 0//n

#define WarpSize 32
#define AutoBlock(a,t) (round)(a/t)+1

unsigned int time_now = 0;

#ifdef __HIPCC__
#define LaunchBound(x,y) __launch_bounds__(x,y)
#else
#define LaunchBound(x,y)
#endif

#if defined(__HIPCC__) // NVCC
#define MY_ALIGN(n) __align__(n)
#elif defined(__GNUC__) // GCC
#define MY_ALIGN(n) __attribute__((aligned(n)))
#elif defined(_MSC_VER) // MSVC
#define MY_ALIGN(n) __declspec(align(n))
#else
#error "Please provide a definition for MY_ALIGN macro for your host compiler!"
#endif

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#pragma region CudaErrorCheck Function
void _check(hipError_t r, int line) {
	if (r != hipSuccess) {
		printf("CUDA error on line %d: %s\n", line, hipGetErrorString(r), line);
		exit(0);
	}
}
#pragma endregion

struct Managed
{
	void *operator new(size_t len) {
		void *ptr;
		CHECK(hipMallocManaged(&ptr, len));
		return ptr;
	}
	void operator delete(void *ptr) {
		CHECK(hipFree(ptr));
	}
};

int CurrentTime = 0;
void PAUSER(int HardStop = 0)
{
	if (0)//HardStop == 4000)//(CurrentTime > 170 && CurrentTime < 172) || HardStop > 8000)//HardStop > (n-2))// || HardStop == 3000)// || (CurrentTime > 170 && CurrentTime < 172) || (CurrentTime > 220))
	{
		printf("\nSystem is PAUSED: Enter 1 to continue:\n");
		int a;
		cin >> a;

		if (a == 0) // exit-code
			exit(0);
	}
}

//////////////////////////////////////////////////////

#include "utility.cuh"

#include "objectiveFunction.cuh"

#include "problem.cuh"

#include "cudaObjects.cuh"

#include "solution.cuh"

//#include "MED_solution.cuh"

#include "bootProblem.cuh"

#include "cudakernels.cuh"

//#include "operators.cuh"
#include "operators_simultaneous.cuh"

//#include "explore_neighborhood.cuh"
#include "explore_neighborhood_simultaneous.cuh"

//#include "lns.cuh"
#include "lns_simultaneous.cuh"

//////////////////////////////////////////////////////

int main()
{
	//File creation for data tracking
	//ofstream out_data("filename.txt");

	printf(".......THE PROGRAM HAS STARTED......\n");
	printf("Original_isGPU=%d, Original_isFull=%d, Original_isRev=%d\n",
			Original_isGPU, Original_isFull, Original_isRev);
	printf("Expectation: %d, ExpectedScenarios: %d\n", Expectation, ExpectedScenarios);

	for (int i = 0; i < 1; i++)
	{

		GenerateScenarios GS;
		GS.Creation();
		GS.Optimize(10000, 6000);
		GS.GPUInformation();
		GS.Destruction();
		CHECK(hipDeviceReset());

		printf("\nABOVE IS THE RUN: %d\n\n", i + 1);
	}
	printf("The Program Ends here..\n");


	return 0;
}
